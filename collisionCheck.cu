#include "hip/hip_runtime.h"
#include "collisionCheck.cuh"

// TODO: fix to only check collision in the spatial dimensions
// 3 has been fixed to be the workspace dimension

__global__ 
void freeEdges(float *obstacles, int obstaclesCount, float *samples, 
	bool *isFreeSamples, int numDisc, float *discMotions, 
	bool *isFreeEdges, int numEdges, float *debugOutput) {
	
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= numEdges)
		return;

	float v[DIM], w[DIM];
	float bbMin[DIM], bbMax[DIM];
	bool motionValid = true;
	for (int i = 0; i < numDisc; ++i) {
		if (!motionValid)
			break;
		int baseIdx = tid*(numDisc+1)*DIM + i*DIM;
		for (int d = 0; d < DIM; ++d) {
			v[d] = discMotions[baseIdx + d];
			w[d] = discMotions[baseIdx + d + DIM];

			if (v[d] > w[d]) {
				bbMin[d] = w[d];
				bbMax[d] = v[d];
			} else {
				bbMin[d] = v[d];
				bbMax[d] = w[d];
			}
		}
		motionValid = motionValid && isMotionValid(v, w, bbMin, bbMax, obstaclesCount, obstacles, debugOutput);
	}

	isFreeEdges[tid] = motionValid;
}

__device__ 
void waypointCollisionCheck(int v_idx, int w_idx, int obstaclesCount, float* obstacles, 
	int *nnIdxs, float *discMotions, int discIdx, int numDisc, bool *isCollision, int tid, float *debugOutput)
{ 
	// motion from w_idx to v_idx
	int discMotionsIdx = nnIdxs[v_idx*NUM + w_idx];

	// calculate bounds of the bounding box
	float v[DIM], w[DIM]; // TODO: do v and w need ot be vectors?
	float bbMin[DIM], bbMax[DIM];
	for (int d = 0; d < DIM; ++d) {
		v[d] = discMotions[discMotionsIdx*(numDisc+1)*DIM + discIdx*DIM + d];
		w[d] = discMotions[discMotionsIdx*(numDisc+1)*DIM + (discIdx+1)*DIM + d];

		if (v[d] > w[d]) {
			bbMin[d] = w[d];
			bbMax[d] = v[d];
		} else {
			bbMin[d] = v[d];
			bbMax[d] = w[d];
		}
	}

	isCollision[tid] = !isMotionValid(v, w, bbMin, bbMax, obstaclesCount, obstacles, debugOutput);
}

__device__
bool isMotionValid(float *v, float *w, float *bbMin, float *bbMax, int obstaclesCount, float* obstacles, float *debugOutput)
{
	// TODO: eventually put each point (v, w) into shared memory
	// TODO: read http://http.developer.nvidia.com/GPUGems3/gpugems3_ch32.html
	// identify which obstacle this processor is checking against

	// I don't think necessary, but routine to check if point is within an obstacle
	// for (int obsIdx = 0; obsIdx < obstaclesCount; ++obsIdx) {
	// 	bool notFree = true;
	// 	for (int d = 0; d < 3; ++d) {
	// 		notFree = notFree && 
	// 		v[d] > obstacles[obsIdx*2*DIM + d] && 
	// 		v[d] < obstacles[obsIdx*2*DIM + DIM + d];
	// 		if (!notFree)
	// 			break;
	// 	}
	// 	if (notFree) {
	// 		return false;
	// 	}
	// }

	// bool same = true;
	// for (int d = 0; d < DIM/2; ++d)
	// 	same = same && (v[d] == w[d]);
	// if (same)
	// 	return true;

	// go through each obstacle and do broad then narrow phase collision checking
	for (int obsIdx = 0; obsIdx < obstaclesCount; ++obsIdx) {
		float obs[DIM*2];
		for (int d = 0; d < DIM; ++d) {
			obs[d] = obstacles[obsIdx*2*DIM + d];
			obs[DIM+d] = obstacles[obsIdx*2*DIM + DIM + d];
		}
		if (!broadphaseValidQ(bbMin, bbMax, obs, debugOutput)) {
			bool motionValid = motionValidQ(v, w, obs, debugOutput);
			if (!motionValid) {
				return false;
			}
		}
	}
	return true;
}

__device__
bool broadphaseValidQ(float *bbMin, float *bbMax, float *obs, float *debugOutput) 
{
	for (int d = 0; d < 3; ++d) {
		if (bbMax[d] <= obs[d] || obs[DIM+d] <= bbMin[d]) 
			return true;
	}
	return false;
}

__device__
bool motionValidQ(float *v, float *w, float *obs, float *debugOutput) 
{
	float v_to_w[3];

	for (int d = 0; d < 3; ++d) {
		float lambda;
		v_to_w[d] = w[d] - v[d];
		if (v[d] < obs[d]) {
			lambda = (obs[d] - v[d])/v_to_w[d];
		} else {
			lambda = (obs[DIM + d] - v[d])/v_to_w[d];
		}
		if (faceContainsProjection(v, w, lambda, d, obs, debugOutput))
			return false;
	}
	return true;
}

__device__
bool faceContainsProjection(float *v, float *w, float lambda, int j, float *obs, 
	float* debugOutput)
{
	for (int d = 0; d < 3; ++d) {
		float projection = v[d] + (w[d] - v[d])*lambda;
		if (d != j && !(obs[d] <= projection && projection <= obs[DIM+d]))
			return false;
	}
	return true;
}

// odd bug when called with v_to_w where the value is not passed correctly 
// resulting in v_to_w[d] = -2e+30 (for example), and collisions being allowed through
// this code is left here to remind me of the error/so I can figure it out later
__device__
bool faceContainsProjectionError(float *v, float *v_to_w, float lambda, int j, float *obs, 
	float* debugOutput)
{
	for (int d = 0; d < 3; ++d) {
		float projection = v[d] + v_to_w[d]*lambda;
		if (d != j && !(obs[d] <= projection && projection <= obs[DIM+d]))
			return false;
	}
	return true;
}

// CPU versions

bool isFreeEdge_h(int edgeIdx, float *obstacles, int obstaclesCount, 
	int numDisc, std::vector<float> discMotions, float *debugOutput) {

	float v[DIM], w[DIM];
	float bbMin[DIM], bbMax[DIM];
	bool motionValid = true;
	for (int i = 0; i < numDisc; ++i) {
		if (!motionValid)
			break;
		int baseIdx = edgeIdx*(numDisc+1)*DIM + i*DIM;
		for (int d = 0; d < DIM; ++d) {
			v[d] = discMotions[baseIdx + d];
			w[d] = discMotions[baseIdx + d + DIM];

			if (v[d] > w[d]) {
				bbMin[d] = w[d];
				bbMax[d] = v[d];
			} else {
				bbMin[d] = v[d];
				bbMax[d] = w[d];
			}
		}
		motionValid = motionValid && isMotionValid_h(v, w, bbMin, bbMax, obstaclesCount, obstacles, debugOutput);
	}

	return motionValid;
}

 
void waypointCollisionCheck_h(int v_idx, int w_idx, int obstaclesCount, float* obstacles, 
	int *nnIdxs, float *discMotions, int discIdx, int numDisc, bool *isCollision, int tid, float *debugOutput)
{ 
	// motion from w_idx to v_idx
	int discMotionsIdx = nnIdxs[v_idx*NUM + w_idx];

	// calculate bounds of the bounding box
	float v[DIM], w[DIM]; // TODO: do v and w need ot be vectors?
	float bbMin[DIM], bbMax[DIM];
	for (int d = 0; d < DIM; ++d) {
		v[d] = discMotions[discMotionsIdx*(numDisc+1)*DIM + discIdx*DIM + d];
		w[d] = discMotions[discMotionsIdx*(numDisc+1)*DIM + (discIdx+1)*DIM + d];

		if (v[d] > w[d]) {
			bbMin[d] = w[d];
			bbMax[d] = v[d];
		} else {
			bbMin[d] = v[d];
			bbMax[d] = w[d];
		}
	}

	isCollision[tid] = !isMotionValid_h(v, w, bbMin, bbMax, obstaclesCount, obstacles, debugOutput);
}

bool isMotionValid_h(float *v, float *w, float *bbMin, float *bbMax, int obstaclesCount, float* obstacles, float *debugOutput)
{
	// TODO: eventually put each point (v, w) into shared memory
	// TODO: read http://http.developer.nvidia.com/GPUGems3/gpugems3_ch32.html
	// identify which obstacle this processor is checking against

	// I don't think necessary, but routine to check if point is within an obstacle
	// for (int obsIdx = 0; obsIdx < obstaclesCount; ++obsIdx) {
	// 	bool notFree = true;
	// 	for (int d = 0; d < 3; ++d) {
	// 		notFree = notFree && 
	// 		v[d] > obstacles[obsIdx*2*DIM + d] && 
	// 		v[d] < obstacles[obsIdx*2*DIM + DIM + d];
	// 		if (!notFree)
	// 			break;
	// 	}
	// 	if (notFree) {
	// 		return false;
	// 	}
	// }
	
	bool same = true;
	for (int d = 0; d < DIM/2; ++d)
		same = same && (v[d] == w[d]);
	if (same)
		return true;

	// go through each obstacle and do broad then narrow phase collision checking
	for (int obsIdx = 0; obsIdx < obstaclesCount; ++obsIdx) {
		float obs[DIM*2];
		for (int d = 0; d < DIM; ++d) {
			obs[d] = obstacles[obsIdx*2*DIM + d];
			obs[DIM+d] = obstacles[obsIdx*2*DIM + DIM + d];
		}
		if (!broadphaseValidQ_h(bbMin, bbMax, obs, debugOutput)) {
			bool motionValid = motionValidQ_h(v, w, obs, debugOutput);
			if (!motionValid) {
				return false;
			}
		}
	}
	return true;
}

bool broadphaseValidQ_h(float *bbMin, float *bbMax, float *obs, float *debugOutput) 
{
	for (int d = 0; d < 3; ++d) {
		if (bbMax[d] <= obs[d] || obs[DIM+d] <= bbMin[d]) 
			return true;
	}
	return false;
}

bool motionValidQ_h(float *v, float *w, float *obs, float *debugOutput) 
{
	float v_to_w[3];

	for (int d = 0; d < 3; ++d) {
		float lambda;
		v_to_w[d] = w[d] - v[d];
		if (v[d] < obs[d]) {
			lambda = (obs[d] - v[d])/v_to_w[d];
		} else {
			lambda = (obs[DIM + d] - v[d])/v_to_w[d];
		}
		if (faceContainsProjection_h(v, w, lambda, d, obs, debugOutput))
			return false;
	}
	return true;
}

bool faceContainsProjection_h(float *v, float *w, float lambda, int j, float *obs, 
	float* debugOutput)
{
	for (int d = 0; d < 3; ++d) {
		float projection = v[d] + (w[d] - v[d])*lambda;
		if (d != j && !(obs[d] <= projection && projection <= obs[DIM+d]))
			return false;
	}
	return true;
}

// odd bug when called with v_to_w where the value is not passed correctly 
// resulting in v_to_w[d] = -2e+30 (for example), and collisions being allowed through
// this code is left here to remind me of the error/so I can figure it out later
bool faceContainsProjectionError_h(float *v, float *v_to_w, float lambda, int j, float *obs, 
	float* debugOutput)
{
	for (int d = 0; d < 3; ++d) {
		float projection = v[d] + v_to_w[d]*lambda;
		if (d != j && !(obs[d] <= projection && projection <= obs[DIM+d]))
			return false;
	}
	return true;
}